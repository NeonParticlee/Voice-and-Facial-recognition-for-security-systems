
#include "CNN.h"

#ifdef __cplusplus
extern "C" {
#endif
#include "Win_FileOps.h"

#ifdef __cplusplus
}
#define sleepEveryNFrames 60 /*will do a modulo operation between a loop incrementer/global counter by this amount and if it equals zero we will invoke Sleep(sleepTime)*/
#define sleepTime 1

void cleanUpRoutine()
{
	dynamicStrCleanUpSession();
	vecClearPointersImgData(&imgData);
	vecClearPointersnormImageData(&normalizedImageDataVec);
	//FileOpsCleanUpSession(); no need right now as no fileops.h operations were used yet
}

int main()
{	
	/*CPU-*/
	defaultPrinterImagePaths = printImagePaths;
	defaultPrinterDirPaths = printDirPaths;
	defaultClearerImgData = clearImgData;
	defaultClearernormImageData = clearNormData;
	
		/*adding new face data folders (each person has their own unique folder in the database)*/
		size_t n = 0;
		char trailing = 0;
		printf("Number of people to add newly: ");
		scanf_s("%zu", &n);
		scanf("%c", &trailing);

		for (size_t i = 0; i < n; ++i)
		{
			char b[MAXPATH];
			printf("enter label: ");
			fgets(b, MAXPATH, stdin);
			for (size_t i = 0; i < MAXPATH; ++i)
			{
				if (b[i] == '\n')
				{
					b[i] = '\0';
					break;
				}
			}
			mkDir(catStrings(DATA_FOLDER_PATH, "\\", b, NULL));
		}
		cleanUpRoutine();

		VectorDirPaths dirs = getDirPaths((char*)DATA_FOLDER_PATH);
		vecPrintDirPaths(&dirs);

		
		VectorImagePaths img = { 0 };
		fetchAllImagesinDirs(&dirs, &img, (char*)".jpg");
		vecPrintImagePaths(&img);
		
		/*---------------------------------IMPORTANT---------------------------------------*
		checkpoint - our file fetching and path construction is done, we can now focus on handling loading the image data using STB to finally process them into the model*/
	/*	for (size_t i = 0; i < img.currEnd; ++i) - train on all images in the whole data folder, too complex right now
		{
			stbImage Image = { 0 };
			loadImage(img.data[i], &Image);
		}
		*/
		stbImage image = { 0 };
		loadImage(img.data[0], &image);
		printf("the path we're working with: %s\n", img.data[0]);
		
		processed_stbImage nImage = { 0 };
		initProcessedImageParams(&nImage, RESIZE_DIMENSIONS, 1);
		preProcessImage(&image, &nImage);
		size_t imgSize = nImage.height * nImage.width * nImage.channels;
		
		/*training session - at the end of each one, various vectors should be reset
		just call cleanUpRoutine() at the end of each session - it takes care of the normalized and non normalized pixel image data along with the dynamicStr function's dynamically allocated strings
		since our file manipulation and image manipulation libraries require a ton of dynamicness and store copies of the heap addresses in those vectors
		other heap address containers like image paths and dirpaths are not exactly critical so they are only cleaned up at the very end of the program
		the fileops header meanwhile is not used yet and will be seldom used - it's just needed to update the models weights at the end of the program or constantly after the end of each full training session on an image/image batch or a testing session if new weight or label data need be stored
		if so just take away the comment of its clearer from the cleanUpRoutine function
		*/







		/*final cleanups; leave at the very end of the code - this block can also be copied to other parts of the code but that won't 
		be really necessary since our path vectors share dynamic memory with utility lib vectors so it is just essential to call the global cleanup function at the end of each 
		major session*/
		/*CPU - end of whole session clean ups*/
		vecClearImagePaths(&img);
		vecClearDirPaths(&dirs);
		cleanUpRoutine();
	return 0;
}


#endif